
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <vector>
#include <iostream>
#include <chrono>

using namespace std;

// adds elements of array in place like this for a 11 element array:
//   [1][1][1][1][1][1][1][1][1][1][1][0][0][0][0][0]
//    ^+=^  ^+=^  ^+=^  ^+=^  ^+=^  ^+=^  ^+=^  ^+=^ 
//   [2][1][2][1][2][1][2][1][2][1][1][0][0][0][0][0]
//    ^ +=  ^     ^ +=  ^     ^ +=  ^     ^ +=  ^       
//   [4][1][2][1][4][1][2][1][3][1][1][0][0][0][0][0]
//    ^    +=     ^           ^    +=     ^                   
//   [8][1][2][1][4][1][2][1][3][1][1][0][0][0][0][0]
//    ^          +=           ^                                           
//   [11][1][2][1][4][1][2][1][3][1][1][0][0][0][0][0]
//    ^ this is the final total
__global__ void addKernel(unsigned int *a, unsigned int interval, unsigned int xDim) {
    unsigned int xInd = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yInd = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int i = interval * (xInd + xDim * yInd);
    a[i] = a[i] + a[i + interval / 2];
}

// Helper function for using CUDA to add vectors in parallel.
unsigned int addWithCuda(unsigned int* aHost, unsigned int size) {
    unsigned int* aDevice;

    // Choose which GPU to run on, change this on a multi-GPU system.
    hipSetDevice(0);

    // Allocate GPU buffers for the vector, ensuring that this buffer is a multiple of 2 equal to or larger than the size of the input array.  set any additional elements to 0
    unsigned int multTwoSize = 2;
    while (multTwoSize < size) {
        multTwoSize = multTwoSize * 2;
    }
    hipMalloc(&aDevice, multTwoSize * sizeof(unsigned int));
    if (multTwoSize > size) {
        hipMemset(&(aDevice[size]), 0, (multTwoSize - size) * sizeof(unsigned int));
    }

    // Copy input vector from host memory to GPU buffer
    hipMemcpy(aDevice, aHost, size * sizeof(unsigned int), hipMemcpyHostToDevice);

    auto start = chrono::high_resolution_clock::now();
    // Launch a kernel on the GPU with one thread first for every other element then every fourth and so on, synchronizing threads after each iteration
    unsigned int interval = 2;
    while (interval <= multTwoSize) {
        unsigned int numThreads = multTwoSize / interval;
        dim3 gridDim(1, 1);
        dim3 blockDim(1, 1);
        // max block dimension is 32x32 threads since the max threads per block is 1024
        // max grid dimension is 2048x2048 blocks assuming each block is 32x32 threads.  This stems from the max x and y dimensions of 65536x65536
        if (numThreads > 32) {
            blockDim.x = 32;
            if ((numThreads / 32) > 32) {
                blockDim.y = 32;
                if ((numThreads / (32 * 32)) > 2048) {
                    gridDim.x = 2048;
                    if ((numThreads / (32 * 32 * 2048)) > 2048) {
                        cout << "Array is too large" << endl;
                        return 0;
                    }
                    else {
                        gridDim.y = numThreads / (32 * 32 * 2048);
                    }
                }
                else {
                    gridDim.x = numThreads / (32 * 32);
                }
            }
            else {
                blockDim.y = numThreads / 32;
            }
        }
        else {
            blockDim.x = numThreads;
        }
        unsigned int xDim = gridDim.x * blockDim.x;
        addKernel <<< gridDim, blockDim >>> (aDevice, interval, xDim);
        hipDeviceSynchronize();
        interval = interval * 2;
    }
    auto end = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::milliseconds>(end - start);
    cout << "Time:  " << duration.count() << " ms" << endl;

    // check for errors during kernel creation
    hipError_t status;
    status = hipGetLastError();
    if (status != hipSuccess) {
        cout << hipGetErrorString(status) << endl;
    }

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(aHost, aDevice, sizeof(unsigned int), hipMemcpyDeviceToHost);

    // free the memory
    hipFree(aDevice);

    // return the total
    unsigned int total = aHost[0];
    return total;
}

int main() {
    vector<unsigned int> a;
    // breaks at 536870913 since this is 2^29 + 1 so multTwo array length will be rounded up to 2^30 = 1073741824 integers times 4 bytes per integer is 4GB which is all of the available GPU memory
    for (unsigned int i = 0; i < 536870912; i++) {
        a.push_back(1);
    }

    // Add elements of the vector in parallel.
    unsigned int total = addWithCuda(&(a[0]), a.size());

    cout << total << endl;

    // cudaDeviceReset must be called before exiting in order for profiling and tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();

    return 0;
}
